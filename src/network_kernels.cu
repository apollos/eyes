#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include <stdio.h>
#include <time.h>
#include <assert.h>

#include "network.h"
#include "image.h"
#include "data.h"
#include "utils.h"
#include "parser.h"

#include "crop_layer.h"
#include "connected_layer.h"
#include "rnn_layer.h"
#include "gru_layer.h"
#include "crnn_layer.h"
#include "detection_layer.h"
#include "convolutional_layer.h"
#include "activation_layer.h"
#include "deconvolutional_layer.h"
#include "maxpool_layer.h"
#include "avgpool_layer.h"
#include "normalization_layer.h"
#include "batchnorm_layer.h"
#include "cost_layer.h"
#include "local_layer.h"
#include "softmax_layer.h"
#include "dropout_layer.h"
#include "route_layer.h"
#include "shortcut_layer.h"
#include "blas.h"
}

float * get_network_output_gpu_layer(network net, int i);
float * get_network_delta_gpu_layer(network net, int i);
float * get_network_output_gpu(network net);

void forward_network_gpu(network net, network_state state)
{
    state.workspace = net.workspace;
    int i;
    for(i = 0; i < net.n; ++i){
        state.index = i;
        layer l = net.layers[i];
        if(l.delta_gpu){
            fill_ongpu(l.outputs * l.batch, 0, l.delta_gpu, 1);
        }
        if(l.type == CONVOLUTIONAL){
            forward_convolutional_layer_gpu(l, state);
        } else if(l.type == DECONVOLUTIONAL){
            forward_deconvolutional_layer_gpu(l, state);
        } else if(l.type == ACTIVE){
            forward_activation_layer_gpu(l, state);
        } else if(l.type == LOCAL){
            forward_local_layer_gpu(l, state);
        } else if(l.type == DETECTION){
            forward_detection_layer_gpu(l, state);
        } else if(l.type == CONNECTED){
            forward_connected_layer_gpu(l, state);
        } else if(l.type == RNN){
            forward_rnn_layer_gpu(l, state);
        } else if(l.type == GRU){
            forward_gru_layer_gpu(l, state);
        } else if(l.type == CRNN){
            forward_crnn_layer_gpu(l, state);
        } else if(l.type == CROP){
            forward_crop_layer_gpu(l, state);
        } else if(l.type == COST){
            forward_cost_layer_gpu(l, state);
        } else if(l.type == SOFTMAX){
            forward_softmax_layer_gpu(l, state);
        } else if(l.type == NORMALIZATION){
            forward_normalization_layer_gpu(l, state);
        } else if(l.type == BATCHNORM){
            forward_batchnorm_layer_gpu(l, state);
        } else if(l.type == MAXPOOL){
            forward_maxpool_layer_gpu(l, state);
        } else if(l.type == AVGPOOL){
            forward_avgpool_layer_gpu(l, state);
        } else if(l.type == DROPOUT){
            forward_dropout_layer_gpu(l, state);
        } else if(l.type == ROUTE){
            forward_route_layer_gpu(l, net);
        } else if(l.type == SHORTCUT){
            forward_shortcut_layer_gpu(l, state);
        }
        state.input = l.output_gpu;
    }
}

void backward_network_gpu(network net, network_state state)
{
    state.workspace = net.workspace;
    int i;
    float * original_input = state.input;
    float * original_delta = state.delta;
    for(i = net.n-1; i >= 0; --i){
        state.index = i;
        layer l = net.layers[i];
        if(i == 0){
            state.input = original_input;
            state.delta = original_delta;
        }else{
            layer prev = net.layers[i-1];
            state.input = prev.output_gpu;
            state.delta = prev.delta_gpu;
        }
        if(l.type == CONVOLUTIONAL){
            backward_convolutional_layer_gpu(l, state);
        } else if(l.type == DECONVOLUTIONAL){
            backward_deconvolutional_layer_gpu(l, state);
        } else if(l.type == ACTIVE){
            backward_activation_layer_gpu(l, state);
        } else if(l.type == LOCAL){
            backward_local_layer_gpu(l, state);
        } else if(l.type == MAXPOOL){
            if(i != 0) backward_maxpool_layer_gpu(l, state);
        } else if(l.type == AVGPOOL){
            if(i != 0) backward_avgpool_layer_gpu(l, state);
        } else if(l.type == DROPOUT){
            backward_dropout_layer_gpu(l, state);
        } else if(l.type == DETECTION){
            backward_detection_layer_gpu(l, state);
        } else if(l.type == NORMALIZATION){
            backward_normalization_layer_gpu(l, state);
        } else if(l.type == BATCHNORM){
            backward_batchnorm_layer_gpu(l, state);
        } else if(l.type == SOFTMAX){
            if(i != 0) backward_softmax_layer_gpu(l, state);
        } else if(l.type == CONNECTED){
            backward_connected_layer_gpu(l, state);
        } else if(l.type == RNN){
            backward_rnn_layer_gpu(l, state);
        } else if(l.type == GRU){
            backward_gru_layer_gpu(l, state);
        } else if(l.type == CRNN){
            backward_crnn_layer_gpu(l, state);
        } else if(l.type == COST){
            backward_cost_layer_gpu(l, state);
        } else if(l.type == ROUTE){
            backward_route_layer_gpu(l, net);
        } else if(l.type == SHORTCUT){
            backward_shortcut_layer_gpu(l, state);
        }
    }
}

void update_network_gpu(network net)
{
    int i;
    int update_batch = net.batch*net.subdivisions;
    float rate = get_current_rate(net);
    for(i = 0; i < net.n; ++i){
        layer l = net.layers[i];
        if(l.type == CONVOLUTIONAL){
            update_convolutional_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == DECONVOLUTIONAL){
            update_deconvolutional_layer_gpu(l, rate, net.momentum, net.decay);
        } else if(l.type == CONNECTED){
            update_connected_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == GRU){
            update_gru_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == RNN){
            update_rnn_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == CRNN){
            update_crnn_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        } else if(l.type == LOCAL){
            update_local_layer_gpu(l, update_batch, rate, net.momentum, net.decay);
        }
    }
}

float train_network_datum_gpu(network net, float *x, float *y)
{
    network_state state;
    state.index = 0;
    state.net = net;
    int x_size = get_network_input_size(net)*net.batch;
    int y_size = get_network_output_size(net)*net.batch;
    if(net.layers[net.n-1].type == DETECTION) y_size = net.layers[net.n-1].truths*net.batch;
    if(!*net.input_gpu){
        *net.input_gpu = cuda_make_array(x, x_size);
        *net.truth_gpu = cuda_make_array(y, y_size);
    }else{
        cuda_push_array(*net.input_gpu, x, x_size);
        cuda_push_array(*net.truth_gpu, y, y_size);
    }
    state.input = *net.input_gpu;
    state.delta = 0;
    state.truth = *net.truth_gpu;
    state.train = 1;
    forward_network_gpu(net, state);
    backward_network_gpu(net, state);
    float error = get_network_cost(net);
    if (((*net.seen) / net.batch) % net.subdivisions == 0) update_network_gpu(net);

    return error;
}

float *get_network_output_layer_gpu(network net, int i)
{
    layer l = net.layers[i];
    cuda_pull_array(l.output_gpu, l.output, l.outputs*l.batch);
    return l.output;
}

float *get_network_output_gpu(network net)
{
    int i;
    for(i = net.n-1; i > 0; --i) if(net.layers[i].type != COST) break;
    return get_network_output_layer_gpu(net, i);
}

float *network_predict_gpu(network net, float *input)
{
    int size = get_network_input_size(net) * net.batch;
    network_state state;
    state.index = 0;
    state.net = net;
    state.input = cuda_make_array(input, size);
    state.truth = 0;
    state.train = 0;
    state.delta = 0;
    forward_network_gpu(net, state);
    float *out = get_network_output_gpu(net);
    cuda_free(state.input);
    return out;
}

