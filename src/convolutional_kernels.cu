#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    binary[i] = (x[i] > 0) ? 1 : -1;
}

void binarize_gpu(float *x, int n, float *binary)
{
    binarize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, binary);
    check_error(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < n; ++i){
        mean += abs(input[i*size + s]);
    }
    mean = mean / n;
    for(i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
    binarize_input_kernel<<<cuda_gridsize(size), BLOCK>>>(input, n, size, binary);
    check_error(hipPeekAtLastError());
}


__global__ void binarize_filters_kernel(float *filters, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
        mean += abs(filters[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (filters[f*size + i] > 0) ? mean : -mean;
    }
}

void binarize_filters_gpu(float *filters, int n, int size, float *binary)
{
    binarize_filters_kernel<<<cuda_gridsize(n), BLOCK>>>(filters, n, size, binary);
    check_error(hipPeekAtLastError());
}

void forward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    int i;
    int m = l.n;
    int k = l.size*l.size*l.c;
    int n = convolutional_out_height(l)*
        convolutional_out_width(l);

    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_filters_gpu(l.filters_gpu, l.n, l.c*l.size*l.size, l.binary_filters_gpu);
        swap_binary(&l);
    }

    if(l.xnor){
        binarize_filters_gpu(l.filters_gpu, l.n, l.c*l.size*l.size, l.binary_filters_gpu);
        swap_binary(&l);
        for(i = 0; i < l.batch; ++i){
            binarize_input_gpu(state.input + i*l.inputs, l.c, l.h*l.w, l.binary_input_gpu + i*l.inputs);
        }
        state.input = l.binary_input_gpu;
    }

#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionForward(cudnn_handle(),
                &one,
                l.srcTensorDesc,
                state.input,
                l.filterDesc,
                l.filters_gpu,
                l.convDesc,
                l.fw_algo,
                state.workspace,
                l.workspace_size,
                &one,
                l.dstTensorDesc,
                l.output_gpu);

#else
    for(i = 0; i < l.batch; ++i){
        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        float * a = l.filters_gpu;
        float * b = state.workspace;
        float * c = l.output_gpu;
        gemm_ongpu(0,0,m,n,k,1.,a,k,b,n,1.,c+i*m*n,n);
    }
#endif

    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, state);
    }
    add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, n);

    activate_array_ongpu(l.output_gpu, m*n*l.batch, l.activation);
    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
}

void backward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    int m = l.n;
    int n = l.size*l.size*l.c;
    int k = convolutional_out_height(l)*
        convolutional_out_width(l);

    gradient_array_ongpu(l.output_gpu, m*k*l.batch, l.activation, l.delta_gpu);

    backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, k);

    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, state);
    }

    if(l.xnor) state.input = l.binary_input_gpu;
#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionBackwardFilter(cudnn_handle(),
            &one,
            l.srcTensorDesc,
            state.input,
            l.ddstTensorDesc,
            l.delta_gpu,
            l.convDesc,
            l.bf_algo,
            state.workspace,
            l.workspace_size,
            &one,
            l.dfilterDesc,
            l.filter_updates_gpu);

    if(state.delta){
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.filterDesc,
                l.filters_gpu,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
                state.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                state.delta);
    }

#else
    int i;
    for(i = 0; i < l.batch; ++i){
        float * a = l.delta_gpu;
        float * b = state.workspace;
        float * c = l.filter_updates_gpu;

        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        gemm_ongpu(0,1,m,n,k,1,a + i*m*k,k,b,k,1,c,n);

        if(state.delta){
            if(l.binary || l.xnor) swap_binary(&l);
            float * a = l.filters_gpu;
            float * b = l.delta_gpu;
            float * c = state.workspace;

            gemm_ongpu(1,0,n,k,m,1,a,n,b + i*k*m,k,0,c,k);

            col2im_ongpu(state.workspace, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.delta + i*l.c*l.h*l.w);
            if(l.binary || l.xnor) swap_binary(&l);
        }
    }
#endif
}

void pull_convolutional_layer(convolutional_layer layer)
{
    cuda_pull_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_pull_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_pull_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_pull_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
}

void push_convolutional_layer(convolutional_layer layer)
{
    cuda_push_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_push_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_push_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_push_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
}

void update_convolutional_layer_gpu(convolutional_layer layer, int batch, float learning_rate, float momentum, float decay)
{
    int size = layer.size*layer.size*layer.c*layer.n;

    axpy_ongpu(layer.n, learning_rate/batch, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.bias_updates_gpu, 1);

    axpy_ongpu(layer.n, learning_rate/batch, layer.scale_updates_gpu, 1, layer.scales_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.scale_updates_gpu, 1);

    axpy_ongpu(size, -decay*batch, layer.filters_gpu, 1, layer.filter_updates_gpu, 1);
    axpy_ongpu(size, learning_rate/batch, layer.filter_updates_gpu, 1, layer.filters_gpu, 1);
    scal_ongpu(size, momentum, layer.filter_updates_gpu, 1);
}


