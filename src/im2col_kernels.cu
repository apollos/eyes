#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "im2col.h"
#include "hip/hip_runtime.h"
}

// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;
                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;
                data_col_ptr += height_col * width_col;
            }
        }
    }
}

void im2col_ongpu(float *im,
         int channels, int height, int width,
         int ksize, int stride, int pad, float *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    pad = pad ? ksize/2 : 0;
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
                width_col, data_col);
}
/*
   __global__ void im2col_pad_kernel(float *im,
   int channels,  int height,  int width,
   int ksize,  int stride, float *data_col)
   {
   int c,h,w;
   int height_col = 1 + (height-1) / stride;
   int width_col = 1 + (width-1) / stride;
   int channels_col = channels * ksize * ksize;

   int pad = ksize/2;

   int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
   int col_size = height_col*width_col*channels_col;
   if (id >= col_size) return;

   int col_index = id;
   w = id % width_col;
   id /= width_col;
   h = id % height_col;
   id /= height_col;
   c = id % channels_col;
   id /= channels_col;

   int w_offset = c % ksize;
   int h_offset = (c / ksize) % ksize;
   int im_channel = c / ksize / ksize;
   int im_row = h_offset + h * stride - pad;
   int im_col = w_offset + w * stride - pad;

   int im_index = im_col + width*(im_row + height*im_channel);
   float val = (im_row < 0 || im_col < 0 || im_row >= height || im_col >= width) ? 0 : im[im_index];

   data_col[col_index] = val;
   }

   __global__ void im2col_nopad_kernel(float *im,
   int channels,  int height,  int width,
   int ksize,  int stride, float *data_col)
   {
   int c,h,w;
   int height_col = (height - ksize) / stride + 1;
   int width_col = (width - ksize) / stride + 1;
   int channels_col = channels * ksize * ksize;

   int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
   int col_size = height_col*width_col*channels_col;
   if (id >= col_size) return;

   int col_index = id;
   w = id % width_col;
   id /= width_col;
   h = id % height_col;
   id /= height_col;
   c = id % channels_col;
   id /= channels_col;

   int w_offset = c % ksize;
   int h_offset = (c / ksize) % ksize;
   int im_channel = c / ksize / ksize;
   int im_row = h_offset + h * stride;
   int im_col = w_offset + w * stride;

   int im_index = im_col + width*(im_row + height*im_channel);
   float val = (im_row < 0 || im_col < 0 || im_row >= height || im_col >= width) ? 0 : im[im_index];

   data_col[col_index] = val;
   }

   extern "C" void im2col_ongpu(float *im,
   int channels,  int height,  int width,
int ksize,  int stride,  int pad, float *data_col)
{

    int height_col = (height - ksize) / stride + 1;
    int width_col = (width - ksize) / stride + 1;
    int channels_col = channels * ksize * ksize;

    if (pad){
        height_col = 1 + (height-1) / stride;
        width_col = 1 + (width-1) / stride;
    }

    size_t n = channels_col*height_col*width_col;

    if(pad)im2col_pad_kernel<<<cuda_gridsize(n),BLOCK>>>(im,  channels, height, width, ksize, stride, data_col);
    else im2col_nopad_kernel<<<cuda_gridsize(n),BLOCK>>>(im,  channels, height, width, ksize, stride, data_col);
    check_error(hipPeekAtLastError());
}
*/
